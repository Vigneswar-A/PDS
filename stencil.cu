#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ int stencil[] = {1, 2, 3};

__global__ void stencil_compute(int* array, int* result, int *n)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	int left = (idx > 0) ? stencil[0]*array[idx-1] : 0;
	int middle = stencil[1]*array[idx];
	int right = (idx < *n-1) ? stencil[2]*array[idx+1] : 0; 
	result[idx] = left+middle+right;
}

int main()
{	
	// create host variables
	int n = 5;
	int array[] = {1, 2, 3, 4, 5};
	int result[n];
	
	// create device variables
	int *dev_n, *dev_array, *dev_result;
	
	// allocate host memory
	hipMalloc(&dev_n, sizeof(int));
	hipMalloc(&dev_array, sizeof(int)*n);
	hipMalloc(&dev_result, sizeof(int)*n);

	// copy input into device memory
	hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_array, &array, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(dev_n, &result, sizeof(int)*n, hipMemcpyHostToDevice);
	
	// call kernel and calculate execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        stencil_compute<<<1, n>>>(dev_array, dev_result, dev_n);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);

	// copy the result to host memory
	hipMemcpy(&result, dev_result, sizeof(int)*n, hipMemcpyDeviceToHost);
	
	// display the result
	printf("Array: ");
	for (int i = 0; i < n; i++)
	{	
		printf("%d ", array[i]);
	}
	printf("\nResult: ");
	for (int i = 0; i < n; i++)
	{
		printf("%d ", result[i]);
	}
	printf("\nExecution Time: %f\n", time);

	// free the device variables
	hipFree(dev_array);
	hipFree(dev_result);
	hipFree(dev_n);

	return 0;
}
