#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dot_product(int *A, int *B, int *C, int *n)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < *n)
	{
		C[idx] = A[idx]*B[idx];
	}
}

__global__ void parallel_reduction(int *C, int *n)
{	
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	for (int stride = *n/2; stride >= 1; stride >>= 1)
	{
		if (idx < *n && idx+stride < *n)
		{
			C[idx] += C[idx+stride];
		}
		__syncthreads();
	}
}

int main()
{	
	// NOTE: n should be power of 2 for the parallel reduction to work correctly!

	// create host variables
	int n = 8;
	int A[] = {1, 2, 3, 4, 5, 6, 7, 8};
	int B[] = {4, 3, 2, 1, 8, 7, 6, 5};
	int C[n];

	// create device variables
	int *dev_A, *dev_B, *dev_C, *dev_n;

	// allocate memory for device variables
	hipMalloc(&dev_A, n*sizeof(int));
	hipMalloc(&dev_B, n*sizeof(int));
	hipMalloc(&dev_C, n*sizeof(int));
	hipMalloc(&dev_n, sizeof(int));

	// tranfer input to device memory
	hipMemcpy(dev_A, &A, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, &B, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
	
	// call kernel and calculate execution time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	dot_product<<<1, n>>>(dev_A, dev_B, dev_C, dev_n);
	parallel_reduction<<<1, n>>>(dev_C, dev_n);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);

	// move result to host variable
	hipMemcpy(&C, dev_C, n*sizeof(int), hipMemcpyDeviceToHost);
	
	// display the result
	printf("Parallel Reduction Result: %d\nElapsed Duration: %f\n", C[0], time);
	
	// free memory of device variables
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	hipFree(dev_n);

	return 0;
}
