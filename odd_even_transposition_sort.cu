#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_transposition_sort(int *array, int *n)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	for (int i = 0; i < *n; i++)
	{
		if (i%2) // odd phase
		{	
			if (idx+1 < *n && idx%2 && array[idx] > array[idx+1])
			{
				int temp = array[idx];
				array[idx] = array[idx+1];
				array[idx+1] = temp;
			}
		}
		else	// even phase
		{
			if (idx+1 < *n && idx%2 == 0 && array[idx] > array[idx+1])
			{
				int temp = array[idx+1];
				array[idx+1] = array[idx];
				array[idx] = temp;
			}
		}
		__syncthreads();
	}
}


int main()
{
	// create host variables
	int array[] = {5, 4, 3, 2, 1, 6, 7, 8, 9, 10};
	int n = sizeof(array)/sizeof(array[0]);

	// create device variables
	int *dev_array, *dev_n;

	// allocate device memory
	hipMalloc(&dev_array, sizeof(int)*n);
	hipMalloc(&dev_n, sizeof(int));
	
	// copy input to device memory
	hipMemcpy(dev_array, &array, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);

	// call kernel and calculate execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        odd_even_transposition_sort<<<1, n>>>(dev_array, dev_n);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);
	
	// copy result to host memory
	hipMemcpy(&array, dev_array, sizeof(int)*n, hipMemcpyDeviceToHost);

	// display the result
	for (int i = 0; i < n; i++)
	{	
		printf("%d ", array[i]);
	}
	printf("\nExecution Time: %f\n", time);

	// free the device memory
	hipFree(dev_array);
	hipFree(dev_n);

	return 0; 
}
