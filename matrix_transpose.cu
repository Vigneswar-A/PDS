#include <hip/hip_runtime.h>
#include <stdio.h>

#define DIM 3

/* 
cost of moving between shared memory < cost of moving between global memory
*/

__global__ void matrix_transpose(int *matrix)
{
	// copy the current element to its correct position on shared matrix
	__shared__ int shared_matrix[DIM][DIM];
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	shared_matrix[idx/DIM][idx%DIM] = matrix[idx];
	
	__syncthreads();
	
	// copy the element to its output location from shared memory
	matrix[idx] = shared_matrix[idx%DIM][idx/DIM];
}

int main()
{
	// create host variables
	int matrix[DIM*DIM] = {1, 2, 3,
			       4, 5, 6,
			       7, 8, 9};

	// create device variables
	int *dev_matrix;

	// allocate device memory
	hipMalloc(&dev_matrix, sizeof(int)*DIM*DIM);
	hipMemcpy(dev_matrix, &matrix, sizeof(int)*DIM*DIM, hipMemcpyHostToDevice);
	
	// display the matrix before transpose
	printf("Before Transpose: \n");
	for (int i = 0; i < DIM*DIM; i++)
	{
		printf("%d ", matrix[i]);
		if ((i+1)%DIM == 0)
		{
			printf("\n");
		}
	}

	// call kernel and calculate execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        matrix_transpose<<<1, DIM*DIM>>>(dev_matrix);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);
	
	// copy result to host memory
	hipMemcpy(&matrix, dev_matrix, sizeof(int)*DIM*DIM, hipMemcpyDeviceToHost);
	
	// display the result
	printf("After Transpose: \n");
	for (int i = 0; i < DIM*DIM; i++)
	{
		printf("%d ", matrix[i]);
		if ((i+1)%DIM == 0)
		{
			printf("\n");
		}
	}
	printf("Transpose Execution Duration: %f\n", time);
	
	// free device memory
	hipFree(dev_matrix);
	return 0; 
}




